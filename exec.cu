#include "hip/hip_runtime.h"
#include "include/defines.h"
#include "include/verify.cuh"
#include "include/winograd.cuh"
#include "include/implgemm.cuh"

#include <stdio.h>

void (*launch_func)(param_t) = launch_implgemm;
// 选择测试用的启动函数类型

int main(int argc, char**argv){
    int n = atoi(argv[1]);
    int c = atoi(argv[2]);
    int h = atoi(argv[3]);
    int w = atoi(argv[4]);
    int k = atoi(argv[5]);
    int r = atoi(argv[6]);
    int s = atoi(argv[7]);
    int u = atoi(argv[8]);
    int v = atoi(argv[9]);
    int p = atoi(argv[10]);
    int q = atoi(argv[11]);

    int outh = (h - r + 2*p)/u + 1;
    int outw = (w - s + 2*q)/v + 1;

    float *pIn       = (float*)malloc(n*c*h*w*sizeof(float));
    float *pWeight   = (float*)malloc(k*c*r*s*sizeof(float));
    float *pOut      = (float*)malloc(n*k*outh*outw*sizeof(float));
    float *pOut_verify = (float*)malloc(n*k*outh*outw*sizeof(float));

    float *pIn_device,*pWeight_device,*pOut_device;
    hipMalloc((void**)&pIn_device, n*c*h*w*sizeof(float));
    hipMalloc((void**)&pWeight_device, k*c*r*s*sizeof(float));
    hipMalloc((void**)&pOut_device, n*k*outh*outw*sizeof(float));

    
    for(int i = 0; i < n*c*h*w; i++){
        pIn[i] = (rand()%255)/255.0;
    }
    
    for(int i = 0; i < k*c*r*s; i++){
        pWeight[i] = (rand()%255)/255.0;
    }
    
    for(int i = 0; i < n*k*outh*outw; i++){
        pOut[i] = 0.0;
        pOut_verify[i] = 0.0;
    }

    hipMemcpy(pIn_device, pIn, n*c*h*w*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(pWeight_device,pWeight,k*c*r*s*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(pOut_device,pOut, n*k*outh*outw*sizeof(float),hipMemcpyHostToDevice);


    /*****************************step 1*****************************/
    param_t param;
    param.in        = pIn_device;        
    param.weight    = pWeight_device;
    param.out       = pOut_device;         
    param.n         = n;                             
    param.c         = c;                             
    param.h         = h;                             
    param.w         = w;                             
    param.k         = k;                             
    param.r         = r;                             
    param.s         = s;                             
    param.u         = u;                             
    param.v         = v;                             
    param.p         = p;                             
    param.q         = q;
    param.Oh = (h - r + 2*p) / u + 1;
    param.Ow = (w - s + 2*q) / v + 1;                               

    /*****************************step 2*****************************/
    int paramSize = sizeof(param_t);
    /*******************************warm up and get result************************************/
    
    launch_verify(param);
    hipMemcpy(pOut_verify, pOut_device,  n*k*outh*outw*sizeof(float), hipMemcpyDeviceToHost);

    launch_func(param);
    hipMemcpy(pOut, pOut_device,  n*k*outh*outw*sizeof(float), hipMemcpyDeviceToHost);

    /*******************************cost time test************************************/

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    
    float time_elapsed = 0.0;
    int iternum = 100;

    for(int i=0; i<iternum; i++){
        launch_func(param);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);
    hipDeviceSynchronize();

    printf("param size :%d \noutput size :%d \ntime: %f us \n",  paramSize, n*k*outh*outw,  time_elapsed * 1000 / iternum);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        hipFree(pIn_device);
        hipFree(pWeight_device);
        hipFree(pOut_device);
        
        free(pIn);
        free(pWeight);
        free(pOut);
        free(pOut_verify);
        exit(-1);
    }
    
    printf("===================start verfiy===================\n");
    

    int error=0;
    for(int i=0;i<n*k*outh*outw;i++){
        if((fabs(pOut[i] - pOut_verify[i]))/ pOut_verify[i] > 0.01 || isnan(pOut[i]) || isinf(pOut[i])){
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, pOut[i], pOut_verify[i]);
            error++;
            break;
        }        
    }

    printf("================finish,error:%d=========================\n",error);

    hipFree(pIn_device);
    hipFree(pWeight_device);
    hipFree(pOut_device);
    
    free(pIn);
    free(pWeight);
    free(pOut);
    free(pOut_verify);
    return 0;
}